#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>


template <typename T>
__global__
void kernel_scan_baseline(T* buffer, int size)
{
    for (int i = 1; i < size; ++i)
        buffer[i] += buffer[i - 1];
}

void baseline_scan(cuda_tools::host_shared_ptr<int> buffer)
{
    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_scan_baseline<int>), hipFuncCachePreferShared);

	kernel_scan_baseline<int><<<1, 1>>>(buffer.data_, buffer.size_);

    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}

template <typename T>
__global__ void kernel_your_scan(const T* __restrict__ g_idata, T* g_odata, int n) {
    extern __shared__ T sdata[];

    int tid = threadIdx.x;
    int pout = 0, pin = 1;

    // exclusive scan so shifted by 1
    sdata[pout * n + tid] = (tid > 0) ? g_idata[tid - 1] : 0;
    __syncthreads();

    for (int offset = 1; offset < n; offset *= 2) {
        pout = 1 - pout;
        pin = 1 - pout;

        if (tid >= offset)
            sdata[pout * n + tid] += sdata[pin * n + tid - offset];
        else
            sdata[pout * n + tid] += sdata[pin * n + tid];
    
        __syncthreads();
    }

    g_odata[tid] += sdata[pout * n + tid];
}

void your_scan(cuda_tools::host_shared_ptr<int> buffer)
{
    hipProfilerStart();

    // TODO
    const int blocksize = 256;
    const int gridsize = (buffer.size_ + blocksize - 1) / blocksize;

    int* index;
    hipMalloc(&index, buffer.size_ * sizeof(int));

    kernel_your_scan<int><<<gridsize, blocksize, buffer.size_ * sizeof(int)>>>(buffer.data_, index, buffer.size_);

    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}