#include "hip/hip_runtime.h"
#include "filter.cuh"

#include <cuda/atomic>

#include "utils.cuh"


__global__
void predicate_kernel(int* predicate_buffer, const int* buffer, const int size) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size)
        predicate_buffer[i] += (buffer[i] != -27);
}


__global__
void scatter_kernel(int* buffer, int* output, const int* predicate, const int size) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < size)
        if (buffer[i] != -27)
            output[predicate[i]] = buffer[i];
}


__global__
void map_kernel(int* buffer, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        int val = buffer[i];
        if (i % 4 == 0) {
            val += 1;
            buffer[i] = max(0, min(255, val));  // Clamp between 0 and 255
        }
        else if (i % 4 == 1) {
            val -= 5;
            buffer[i] = max(0, min(255, val));
        }
        else if (i % 4 == 2) {
            val += 3;
            buffer[i] = max(0, min(255, val));
        }
        else if (i % 4 == 3) {
            val -= 8;
            buffer[i] = max(0, min(255, val));
        }
    }
}

template <FindType F>
__global__
void find_first_value(const int *data, const int size, const int valueToFind, int *result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size) {
        if constexpr (F == FindType::SMALLER) {
            if (data[tid] < valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
        else if constexpr (F == FindType::EQUAL) {
            if (data[tid] == valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
        else if constexpr (F == FindType::BIGGER) {
            if (data[tid] > valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
    }
}


void predicate(int* predicate, const int* buffer, const int size, hipStream_t& stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    predicate_kernel<<<grid_size, block_size, 0, stream>>>(predicate, buffer, size);
}


void scatter(int* buffer, int* output, const int* predicate, const int size, hipStream_t& stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    scatter_kernel<<<grid_size, block_size, 0, stream>>>(buffer, output, predicate, size);
}


void map(int* buffer, const int size, hipStream_t& stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    map_kernel<<<grid_size, block_size, 0, stream>>>(buffer, size);
}

template <FindType F>
int find_index(const int* buffer, const int size, const int value, hipStream_t& stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    int* result;
    CUDA_CALL(hipMallocManaged(&result, sizeof(int)));
    CUDA_CALL(hipMemset(result, -1, sizeof(int)));

    find_first_value<F><<<grid_size, block_size, 0, stream>>>(buffer, size, value, result);
    CUDA_CALL(hipStreamSynchronize(stream));

    int res = *result;
    CUDA_CALL(hipFree(result));

    return res;
}

template int find_index<FindType::SMALLER>(const int*, const int, const int, hipStream_t&);
template int find_index<FindType::EQUAL>(const int*, const int, const int, hipStream_t&);
template int find_index<FindType::BIGGER>(const int*, const int, const int, hipStream_t&);
