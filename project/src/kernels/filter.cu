#include "hip/hip_runtime.h"
#include "filter.cuh"

#include "utils.cuh"


__global__
void predicate_kernel(int* predicate, const int* buffer, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
        predicate[i] = buffer[i] != -27 ? 1 : 0;
}


__global__
void scatter_kernel(int* buffer, const int* predicate, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && buffer[i] != -27)
        buffer[predicate[i]] = buffer[i];
}


__global__
void map_kernel(int* buffer, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        if (i % 4 == 0)
            buffer[i] += 1;
        else if (i % 4 == 1)
            buffer[i] -= 5;
        else if (i % 4 == 2)
            buffer[i] += 3;
        else if (i % 4 == 3)
            buffer[i] -= 8;
    }
}


void predicate(int* predicate, const int* buffer, const int size, hipStream_t* stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    predicate_kernel<<<grid_size, block_size, 0, *stream>>>(predicate, buffer, size);

    CUDA_CALL(hipDeviceSynchronize());
}


void scatter(int* buffer, const int* predicate, const int size, hipStream_t* stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    scatter_kernel<<<grid_size, block_size, 0, *stream>>>(buffer, predicate, size);

    CUDA_CALL(hipDeviceSynchronize());
}


void map(int* buffer, const int size, hipStream_t* stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    map_kernel<<<grid_size, block_size, 0, *stream>>>(buffer, size);

    CUDA_CALL(hipDeviceSynchronize());
}
