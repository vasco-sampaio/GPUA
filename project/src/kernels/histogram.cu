#include "hip/hip_runtime.h"
#include "histogram.cuh"

#include <cuda/atomic>

#include "utils.h"


__global__
void histogram_kernel(int* histogram, const int* buffer, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
        atomicAdd(&histogram[buffer[i]], 1);
}


__global__
void histogram_equalization_kernel(int* buffer, const int* histogram, const int size, const int min, const int max) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
        buffer[i] = (buffer[i] - min) * (max - min) / (size - 1);
}


void histogram(int* histogram, const int* buffer, const int size) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    hipMemset(histogram, 0, 256 * sizeof(int));

    histogram_kernel<<<grid_size, block_size, 0>>>(histogram, buffer, size);


}


void histogram_equalization(int* buffer, const int* histogram, const int size, const int min, const int max) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    histogram_equalization_kernel<<<grid_size, block_size, 0>>>(buffer, histogram, size, min, max);


}
