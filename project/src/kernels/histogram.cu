#include "hip/hip_runtime.h"
#include "histogram.cuh"

#include <cuda/atomic>

#include "utils.cuh"


__global__
void histogram_kernel(int* histogram, const int* buffer, const int size) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
        atomicAdd(&histogram[buffer[i]], 1);
}


__global__
void histogram_equalization_kernel(int* buffer, const int* histogram, const int size, const int min, const int max) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
        buffer[i] = (buffer[i] - min) * (max - min) / (size - 1);
}


void histogram(int* histogram, const int* buffer, const int size, hipStream_t* stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    CUDA_CALL(hipMemset(histogram, 0, 256 * sizeof(int)));

    histogram_kernel<<<grid_size, block_size, 0, *stream>>>(histogram, buffer, size);

    CUDA_CALL(hipDeviceSynchronize());
}


void histogram_equalization(int* buffer, const int* histogram, const int size, const int min, const int max, hipStream_t* stream) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    histogram_equalization_kernel<<<grid_size, block_size, 0, *stream>>>(buffer, histogram, size, min, max);

    CUDA_CALL(hipDeviceSynchronize());
}
