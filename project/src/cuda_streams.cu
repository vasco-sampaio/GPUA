#include "cuda_streams.cuh"

hipStream_t streams[NUM_STREAMS];

void initializeStreams() {
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
    }
}

hipStream_t getStream(int index) {
    return streams[index];
}

void cleanupStreams() {
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
}