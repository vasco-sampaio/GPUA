#include "cuda_streams.cuh"

#include "utils.cuh"

hipStream_t streams[NUM_STREAMS];

void initializeStreams() {
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CALL(hipStreamCreate(&streams[i]));
    }
}

hipStream_t getStream(int index) {
    return streams[index];
}

void cleanupStreams() {
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CALL(hipStreamSynchronize(streams[i]));
        CUDA_CALL(hipStreamDestroy(streams[i]));
    }
}