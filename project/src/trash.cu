
#include <hip/hip_runtime.h>
// Padding the input array to the next power of 2 --> BAD IDEA
//void scan(int* input, int* output, int n) {
//    long pow_2 = NEXT_POW_2(n);
//    if (n == pow_2) {
//        scan_pow_2(input, output, n);
//        return;
//    }
//
//    int prev_pow_2 = PREV_POW_2(n);
//
//    if (n - prev_pow_2 < pow_2 - n) {
//        scan_pow_2(input, output, prev_pow_2);
//
//        int *last = output + prev_pow_2 - 1;
//
//        int rem = n - prev_pow_2;
//        int n_rem = NEXT_POW_2(rem + 1);
//
//        int *rem_ptr;
//        int *output_rem;
//        CUDA_CALL(cudaMalloc(&output_rem, n_rem * sizeof(int)));
//        CUDA_CALL(cudaMalloc(&rem_ptr, n_rem * sizeof(int)));
//
//        CUDA_CALL(cudaMemcpy(rem_ptr, last, sizeof(int), cudaMemcpyDeviceToDevice));
//        CUDA_CALL(cudaMemcpy(rem_ptr + 1, input + prev_pow_2, rem * sizeof(int), cudaMemcpyDeviceToDevice));
//        CUDA_CALL(cudaMemset(rem_ptr + 1 + rem , 0, (n_rem - rem - 1) * sizeof(int)));
//
//        scan_pow_2(rem_ptr, output_rem, n_rem);
//
//        CUDA_CALL(cudaMemcpy(output + prev_pow_2, output_rem + 1, rem * sizeof(int), cudaMemcpyDeviceToDevice));
//
//        CUDA_CALL(cudaFree(rem_ptr));
//        CUDA_CALL(cudaFree(output_rem));
//    } else {
//        int *input_pow_2, *output_pow_2;
//
//        CUDA_CALL(cudaMalloc(&input_pow_2, pow_2 * sizeof(int)));
//        CUDA_CALL(cudaMalloc(&output_pow_2, pow_2 * sizeof(int)));
//
//        CUDA_CALL(cudaMemcpy(input_pow_2, input, n * sizeof(int), cudaMemcpyDeviceToDevice));
//        CUDA_CALL(cudaMemset(input_pow_2 + n, 0, (pow_2 - n) * sizeof(int)));
//
//        scan_pow_2(input_pow_2, output_pow_2, pow_2);
//
//        CUDA_CALL(cudaMemcpy(output, output_pow_2, n * sizeof(int), cudaMemcpyDeviceToDevice));
//
//        CUDA_CALL(cudaFree(input_pow_2));
//        CUDA_CALL(cudaFree(output_pow_2));
//    }
//}