#include "hip/hip_runtime.h"
// Padding the input array to the next power of 2 --> BAD IDEA
//void scan(int* input, int* output, int n) {
//    long pow_2 = NEXT_POW_2(n);
//    if (n == pow_2) {
//        scan_pow_2(input, output, n);
//        return;
//    }
//
//    int prev_pow_2 = PREV_POW_2(n);
//
//    if (n - prev_pow_2 < pow_2 - n) {
//        scan_pow_2(input, output, prev_pow_2);
//
//        int *last = output + prev_pow_2 - 1;
//
//        int rem = n - prev_pow_2;
//        int n_rem = NEXT_POW_2(rem + 1);
//
//        int *rem_ptr;
//        int *output_rem;
//        CUDA_CALL(hipMalloc(&output_rem, n_rem * sizeof(int)));
//        CUDA_CALL(hipMalloc(&rem_ptr, n_rem * sizeof(int)));
//
//        CUDA_CALL(hipMemcpy(rem_ptr, last, sizeof(int), hipMemcpyDeviceToDevice));
//        CUDA_CALL(hipMemcpy(rem_ptr + 1, input + prev_pow_2, rem * sizeof(int), hipMemcpyDeviceToDevice));
//        CUDA_CALL(hipMemset(rem_ptr + 1 + rem , 0, (n_rem - rem - 1) * sizeof(int)));
//
//        scan_pow_2(rem_ptr, output_rem, n_rem);
//
//        CUDA_CALL(hipMemcpy(output + prev_pow_2, output_rem + 1, rem * sizeof(int), hipMemcpyDeviceToDevice));
//
//        CUDA_CALL(hipFree(rem_ptr));
//        CUDA_CALL(hipFree(output_rem));
//    } else {
//        int *input_pow_2, *output_pow_2;
//
//        CUDA_CALL(hipMalloc(&input_pow_2, pow_2 * sizeof(int)));
//        CUDA_CALL(hipMalloc(&output_pow_2, pow_2 * sizeof(int)));
//
//        CUDA_CALL(hipMemcpy(input_pow_2, input, n * sizeof(int), hipMemcpyDeviceToDevice));
//        CUDA_CALL(hipMemset(input_pow_2 + n, 0, (pow_2 - n) * sizeof(int)));
//
//        scan_pow_2(input_pow_2, output_pow_2, pow_2);
//
//        CUDA_CALL(hipMemcpy(output, output_pow_2, n * sizeof(int), hipMemcpyDeviceToDevice));
//
//        CUDA_CALL(hipFree(input_pow_2));
//        CUDA_CALL(hipFree(output_pow_2));
//    }
//}

__device__
int scan_warp(int* data, const int tid) {
    const int lane = tid & 31; // index within the warp

    if (lane >= 1) data[tid] += data[tid - 1]; __syncwarp();
    if (lane >= 2) data[tid] += data[tid - 2]; __syncwarp();
    if (lane >= 4) data[tid] += data[tid - 4]; __syncwarp();
    if (lane >= 8) data[tid] += data[tid - 8]; __syncwarp();
    if (lane >= 16) data[tid] += data[tid - 16]; __syncwarp();

    return data[tid];
}


__device__
int scan_block(int* data, const int tid) {
    const int lane = tid & 31;
    const int warpid = tid >> 5;

    int val = scan_warp(data, tid);
    __syncthreads();

    if(lane == 31) data[warpid] = data[tid]; __syncthreads();

    if(warpid == 0) {
        scan_warp(data, tid); __syncthreads();
        printf("Two first values of the block %d: %d %d\n", blockIdx.x, data[0], data[1]);
    }

    if (warpid > 0) val += data[warpid - 1]; __syncthreads();

    data[tid] = val;
    __syncthreads();

    return val;
}

template <FindType F>
__global__
void find_first_value(const int *data, const int size, const int valueToFind, int *result) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < size) {
        if constexpr (F == FindType::SMALLER) {
            if (data[tid] < valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
        else if constexpr (F == FindType::EQUAL) {
            if (data[tid] == valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
        else if constexpr (F == FindType::BIGGER) {
            if (data[tid] > valueToFind) {
                if (atomicCAS(result, -1, tid) != -1)
                    atomicMin(result, tid);
            }
        }
    }
}


template <FindType F>
int find_index(const int* buffer, const int size, const int value) {
    const int block_size = BLOCK_SIZE(size);
    const int grid_size = (size + block_size - 1) / block_size;

    int* result;
    CUDA_CALL(hipMalloc(&result, sizeof(int)));
    CUDA_CALL(hipMemsetAsync(result, -1, sizeof(int)));

    find_first_value<F><<<grid_size, block_size>>>(buffer, size, value, result);

    CUDA_CALL(hipStreamSynchronize(stream));

    int *tmp;
    CUDA_CALL(hipHostMalloc(&tmp, sizeof(int)));
    CUDA_CALL(hipMemcpyAsync(tmp, result, sizeof(int), hipMemcpyDeviceToHost));

    int res = *tmp;

    CUDA_CALL(hipHostFree(tmp));
    CUDA_CALL(hipFree(result));

    return res;
}

template int find_index<FindType::SMALLER>(const int*, const int, const int, hipStream_t&);
template int find_index<FindType::EQUAL>(const int*, const int, const int, hipStream_t&);
template int find_index<FindType::BIGGER>(const int*, const int, const int, hipStream_t&);

enum class FindType {
    SMALLER,
    EQUAL,
    BIGGER
};

void check_predicate(int* d_buffer, int* d_predicate, int size){
    std::vector<int> h_buffer(size, 0);
    hipMemcpy(h_buffer.data(), d_buffer, size*sizeof(int), hipMemcpyDeviceToHost);
    std::vector<int> gpu_predicate(size, 0);
    hipMemcpy(gpu_predicate.data(), d_predicate, size*sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> cpu_predicate(size, 0);
    constexpr int garbage_val = -27;
    int count_garbage = 0;
    for (int i = 0; i < size; ++i) {
        if (h_buffer[i] != garbage_val)
            cpu_predicate[i] = 1;
        else
            count_garbage++;
    }

    printf("cpu_size: %lu, gpu_size: %lu, garbage_count: %d\n", cpu_predicate.size(), gpu_predicate.size(), count_garbage);

    bool same = true;
    int count = 0;
    for (int i = 0; i < size; i++){
        if (cpu_predicate[i] != gpu_predicate[i]){
            same = false;
            count++;
            //printf("index: %d, cpu: %d, gpu: %d\n", i, cpu_predicate[i], gpu_predicate[i]);
        }
    }

    if (same)
        printf("predicate good !\n");
    else
        printf("predicate bad !, %d are bad, %i\n", count, (count/size)*100);
}

void check_scan(int* d_predicate, int* d_scan_result, int size){
    std::vector<int> h_scan_result(size, 0);
    hipMemcpy(h_scan_result.data(), d_scan_result, size*sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> h_predicate(size, 0);
    hipMemcpy(h_predicate.data(), d_predicate, size * sizeof(int), hipMemcpyDeviceToHost);
    std::exclusive_scan(h_predicate.begin(), h_predicate.end(), h_predicate.begin(), 0);


    bool same = true;
    int count = 0;
    for (int i = 0; i < size; i++){
        if (h_predicate[i] != h_scan_result[i]){
            same = false;
            count++;
            //printf("index: %d, cpu: %d, gpu: %d\n", i, h_predicate[i], h_scan_result[i]);
        }
    }

    if (same)
        printf("scan good !\n");
    else
        printf("scan bad !, %d are bad, %i\n", count, (count/size)*100);

}

void check_scatter(int *my_d_buffer, int *d_buffer, int *d_predicate, int size, int compact_size){
    std::vector<int> h_buffer(size, 0);
    hipMemcpy(h_buffer.data(), d_buffer, size*sizeof(int), hipMemcpyDeviceToHost);
    std::vector<int> my_h_buffer(size, 0);
    hipMemcpy(my_h_buffer.data(), my_d_buffer, size*sizeof(int), hipMemcpyDeviceToHost);
    std::vector<int> h_predicate(size, 0);
    hipMemcpy(h_predicate.data(), d_predicate, size*sizeof(int), hipMemcpyDeviceToHost);

    constexpr int garbage_val = -27;
    for (std::size_t i = 0; i < size; ++i) {
        if (h_buffer[i] != garbage_val) {
            h_buffer[h_predicate[i]] = h_buffer[i];
        }
    }

    bool same = true;
    int count = 0;
    for (int i = 0; i < size; i++){
        if (h_buffer[i] != my_h_buffer[i]){
            same = false;
            count++;
            //printf("index: %d, cpu: %d, gpu: %d\n", i, h_buffer[i], my_h_buffer[i]);
        }
    }

    if (same)
        printf("scatter good !\n");
    else
        printf("scatter bad !, %d are bad, %f\n", count, ((float)count/size)*100);
}
void check_histogram(int* d_histogram, int* d_buffer, int histogram_size, int image_size){
    std::vector<int> h_histogram(histogram_size, 0);
    hipMemcpy(h_histogram.data(), d_histogram, histogram_size*sizeof(int), hipMemcpyDeviceToHost);
    std::vector<int> h_buffer(image_size, 0);
    hipMemcpy(h_buffer.data(), d_buffer, image_size*sizeof(int), hipMemcpyDeviceToHost);

    std::array<int, 256> histo;
    histo.fill(0);
    for (int i = 0; i < image_size; ++i)
        ++histo[h_buffer[i]];

    bool same = true;
    int count = 0;
    for (int i = 0; i < histogram_size; i++){
        if (histo[i] != h_histogram[i]){
            same = false;
            count++;
            //printf("index: %d, cpu: %d, gpu: %d\n", i, h_buffer[i], my_h_buffer[i]);
        }
    }

    if (same)
        printf("calculate histogram good !\n");
    else
        printf("calculate histogram !, %d are bad, %f\n", count, ((float)count/histogram_size)*100);
}
