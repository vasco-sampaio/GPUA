#include "image.hh"
#include "pipeline.hh"
#include "fix_cpu.cuh"
#include "fix_gpu.cuh"
#include "kernels/scan.cuh"
#include "kernels/reduce.cuh"

#include <vector>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <filesystem>
#include <numeric>


int main([[maybe_unused]] int argc, [[maybe_unused]] char* argv[])
{
    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    using recursive_directory_iterator = std::filesystem::recursive_directory_iterator;
    std::vector<std::string> filepaths;
    for (const auto& dir_entry : recursive_directory_iterator("/afs/cri.epita.fr/resources/teach/IRGPUA/images"))
        filepaths.emplace_back(dir_entry.path());


    // - Init pipeline object

    Pipeline pipeline(filepaths);

    // -- Main loop containing image retrying from pipeline and fixing

    const int nb_images = pipeline.images.size();
    std::cout << "Nb images: " << nb_images << std::endl;
    std::vector<Image> images(nb_images);

    std::cout << "Done, starting compute" << std::endl;

    // -- Pipeline initialization

    std::cout << "File loading..." << std::endl;

    // - Get file paths

    // #pragma omp parallel for
    for (int i = 0; i < nb_images; ++i)
    {
        images[i] = pipeline.get_image(i);
        fix_image_gpu(images[i]);
    }

    std::cout << "Done with compute, starting stats" << std::endl;

    for (int i = 0; i < nb_images; ++i)
        hipHostFree(images[i].buffer);

    return 0;
}
