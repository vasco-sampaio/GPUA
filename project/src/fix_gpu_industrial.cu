#include "fix_gpu_industrial.cuh"

#include "CUB_Thrust/compact.cuh"
#include "CUB_Thrust/map.cuh"
#include "CUB_Thrust/histogram.cuh"


void fix_gpu_industrial(Image& image) {
    const int buffer_size = image.size();
    const int image_size = image.width * image.height;

    // #1 Compact
    int* d_input;
    int *d_output;
    hipMalloc(&d_input, buffer_size * sizeof(int));
    hipMalloc(&d_output, buffer_size * sizeof(int));

    hipMemcpy(d_input, image.buffer, buffer_size * sizeof(int), hipMemcpyHostToDevice);

    compact_scan(d_input, d_output, buffer_size);

    // #2 Apply map to fix pixels
    modify_buffer(d_output, d_output, image_size);

    // #3 Histogram
    int* histogram;
    hipMalloc(&histogram, 257 * sizeof(int)); // CUB requires 257 bins, otherwise the last bin is not computed
    hipMemset(histogram, 0, 257 * sizeof(int));

    histogram_equalization(d_output, histogram, image_size);

    hipMemcpy(image.buffer, d_output, image_size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
