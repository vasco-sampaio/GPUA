#include "fix_gpu.cuh"

#include "cuda_streams.cuh"
#include "kernels/filter.cuh"
#include "kernels/histogram.cuh"
#include "kernels/scan.cuh"
#include "kernels/utils.cuh"


void fix_image_gpu(Image& image, hipStream_t& stream) {
    const int image_size = image.width * image.height;
    const int buffer_size = image.size();

    int* d_buffer;
    int* d_predicate_buffer;
    int* d_histo;

    CUDA_CALL(hipMalloc(&d_buffer, buffer_size * sizeof(int)));
    CUDA_CALL(hipMalloc(&d_predicate_buffer, buffer_size * sizeof(int)));

    CUDA_CALL(hipMemcpyAsync(d_buffer, image.buffer, buffer_size * sizeof(int), hipMemcpyHostToDevice, stream));
    CUDA_CALL(hipMemsetAsync(d_predicate_buffer, 0, buffer_size * sizeof(int), stream));

    predicate(d_predicate_buffer, d_buffer, buffer_size, stream);
    scan<ScanType::EXCLUSIVE>(d_predicate_buffer, d_predicate_buffer, buffer_size, stream);
    scatter(d_buffer, d_buffer, d_predicate_buffer, buffer_size, stream);
    
    CUDA_CALL(hipFree(d_predicate_buffer)); // stuck here


    map(d_buffer, image_size, stream);

    CUDA_CALL(hipMalloc(&d_histo, 256 * sizeof(int)));
    CUDA_CALL(hipMemsetAsync(d_histo, 0, 256 * sizeof(int), stream));

    histogram(d_histo, d_buffer, image_size, stream);
    scan<ScanType::INCLUSIVE>(d_histo, d_histo, 256, stream);

    int host_histo[256];
    CUDA_CALL(hipMemcpyAsync(host_histo, d_histo, 256 * sizeof(int), hipMemcpyDeviceToHost, stream));

    int* first_non_zero = std::find_if(host_histo, host_histo + 256, [](int val) { return val != 0; });
    histogram_equalization(d_buffer, d_histo, image_size, *first_non_zero, stream);

    CUDA_CALL(hipMemcpyAsync(image.buffer, d_buffer, image_size * sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CALL(hipFree(d_buffer));
    CUDA_CALL(hipFree(d_histo));
}