#include "fix_gpu.cuh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <iostream>

#include "kernels/scan.cuh"
#include "kernels/filter.cuh"
#include "kernels/histogram.cuh"


#include <iostream>

void fix_image_gpu(int* buffer, const int buffer_size, const int image_size, hipStream_t* stream)
{
    int* predicate_buffer;
    hipMalloc(&predicate_buffer, buffer_size * sizeof(int));
    hipMemset(predicate_buffer, 0, buffer_size * sizeof(int));

    predicate(predicate_buffer, buffer, buffer_size, stream);

    scan(predicate_buffer, predicate_buffer, buffer_size, stream, false);

    int* image_buffer;
    hipMalloc(&image_buffer, image_size * sizeof(int));

    scatter(buffer, image_buffer, predicate_buffer, buffer_size, stream);

    hipFree(predicate_buffer);

    hipMemcpyAsync(buffer, image_buffer, image_size * sizeof(int), hipMemcpyDeviceToDevice, *stream);

    hipFree(image_buffer);
}
