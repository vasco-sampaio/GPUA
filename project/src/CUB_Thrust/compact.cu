#include "compact.cuh"

#include <hipcub/hipcub.hpp>


struct DifferentThan
{
    int compare;

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    DifferentThan(int compare) : compare(compare) {}

    HIPCUB_RUNTIME_FUNCTION __forceinline__
    bool operator()(const int &a) const {
        return (a != compare);
    }
};

void compact_scan(int* d_input, int* d_output, const int size)
{
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;
    int      *d_num_selected_out = nullptr;

    hipMalloc(&d_num_selected_out, sizeof(int));
    hipMemset(d_num_selected_out, 0, sizeof(int));

    DifferentThan select_op(-27);

    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_input, d_output, d_num_selected_out, size, select_op);

    hipMalloc(&d_temp_storage, temp_storage_bytes);
  
    hipcub::DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_input, d_output, d_num_selected_out, size, select_op);

    hipFree(d_temp_storage);
    hipFree(d_num_selected_out);
}