#include "hip/hip_runtime.h"
#include "to_bench.cuh"

#include "cuda_tools/cuda_error_checking.cuh"
#include "cuda_tools/host_shared_ptr.cuh"

#include <hip/hip_runtime_api.h>

#include <iostream>

template <typename T>
__global__
void kernel_reduce_baseline(const T* __restrict__ buffer, T* __restrict__ total, int size)
{
    const int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < size)
        atomicAdd(&total[0], buffer[id]);
}

void baseline_reduce(cuda_tools::host_shared_ptr<int> buffer,
    cuda_tools::host_shared_ptr<int> total)
{
    hipProfilerStart();
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_reduce_baseline<int>), hipFuncCachePreferShared);

    constexpr int blocksize = 64;
    const int gridsize = (buffer.size_ + blocksize - 1) / blocksize;

    kernel_reduce_baseline<int><<<gridsize, blocksize>>>(buffer.data_, total.data_, buffer.size_);

    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}

template <typename T>
__global__
void kernel_your_reduce(const T* __restrict__ buffer, T* __restrict__ total, int size)
{
    // TODO
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    sdata[tid] = buffer[i]+ buffer[i + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        total[blockIdx.x] = sdata[0];
}

void your_reduce(cuda_tools::host_shared_ptr<int> buffer,
    cuda_tools::host_shared_ptr<int> total)
{
    hipProfilerStart();

    /*
     * Si le nombre de threads est trop faible on va generer un grand nombre de blocs et le
     * deuxieme appel au kernel ne va pas traiter toutes les valeurs car pas assez de threads dans un bloc
     */ 
    const int blockSize = 1024;
    /* 
     * On divise le nombre de blocs par 2 parce que chaque thread est desormais charge de load 2 valeurs, 
     * ceci a ete fait parce que au bout d'une iteration la moitie des threads etait inutilisee donc autant 
     * augmenter le work per thread
     */
    const int gridSize = (buffer.size_ + blockSize - 1) / (blockSize * 2); 

    int *tmp;
    hipMalloc(&tmp, gridSize * sizeof(int));

    kernel_your_reduce<int><<<gridSize, blockSize, sizeof(int) * blockSize>>>(buffer.data_, tmp, blockSize);

    kernel_your_reduce<int><<<1, gridSize, sizeof(int) * gridSize>>>(tmp, total.data_, gridSize);
    
    hipDeviceSynchronize();
    kernel_check_error();
    
    hipProfilerStop();
}